#include "MatrixMath.cuh"
#include "cuda/commoncu.cuh"
#include <hipfft/hipfft.h>
#include <cutil_inline.h>

namespace eagleeye
{
void convolution2DCu(const float* src,
	const unsigned int src_rows,
	const unsigned int src_cols,
	const float* kernel,
	const unsigned int kernel_rows,
	const unsigned int kernel_cols,float* result)
{
	const int kernel_h=kernel_rows;
	const int kernel_w=kernel_cols;
	const int data_h=src_rows;
	const int data_w=src_cols;
	const int fft_h=snapTransformSize(data_h+kernel_h-1);
	const int fft_w=snapTransformSize(data_w+kernel_w-1);

	float* h_result;
	h_result=(float*)malloc(sizeof(float)*fft_h*fft_w);

	float* d_src;
	float* d_kernel;

	float* d_padded_src;
	float* d_padded_kernel;

	fComplex* d_src_spectrum;
	fComplex* d_kernel_spectrum;

	cutilSafeCall(hipMalloc((void**)&d_src,data_h*data_w*sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_kernel,kernel_h*kernel_w*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_padded_src,fft_h*fft_w*sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_padded_kernel,fft_h*fft_w*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_src_spectrum,fft_h * (fft_w / 2 + 1)*sizeof(fComplex)));
	cutilSafeCall(hipMalloc((void**)&d_kernel_spectrum,fft_h * (fft_w / 2 + 1)*sizeof(fComplex)));

	cutilSafeCall(hipMemcpy(d_kernel,kernel,kernel_h*kernel_w*sizeof(float),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_src,src,data_h*data_w*sizeof(float),hipMemcpyHostToDevice));

	cutilSafeCall(hipMemset(d_padded_kernel,0,fft_w*fft_h*sizeof(float)));
	cutilSafeCall(hipMemset(d_padded_src,0,fft_w*fft_h*sizeof(float)));

	padData(d_padded_src,fft_h,fft_w,d_src,data_h,data_w,kernel_h/2,kernel_w/2);
	padData(d_padded_kernel,fft_h,fft_w,d_kernel,kernel_h,kernel_w);

	hipfftHandle fftplan_fwd,fftplan_inv;
	cufftSafeCall(hipfftPlan2d(&fftplan_fwd,fft_h,fft_w,HIPFFT_R2C));
	cufftSafeCall(hipfftPlan2d(&fftplan_inv,fft_h,fft_w,HIPFFT_C2R));

	//running GPU FFT convolution
	cufftSafeCall(hipfftExecR2C(fftplan_fwd,(hipfftReal*)d_padded_src,(hipfftComplex*)d_src_spectrum));
	cufftSafeCall(hipfftExecR2C(fftplan_fwd,(hipfftReal*)d_padded_kernel,(hipfftComplex*)d_kernel_spectrum));

	cutilSafeCall(cutilDeviceSynchronize());
	conjugate(d_kernel_spectrum,fft_h * (fft_w / 2 + 1));
	multiplyAndScale(d_src_spectrum,d_kernel_spectrum,1.0f/(fft_h*fft_w),d_src_spectrum,fft_h * (fft_w / 2 + 1));


	cufftSafeCall(hipfftExecC2R(fftplan_inv,(hipfftComplex*)d_src_spectrum,(hipfftReal*)d_padded_src));
	cutilSafeCall(cutilDeviceSynchronize());

	cutilSafeCall(hipMemcpy(h_result,d_padded_src,fft_h*fft_w*sizeof(float),hipMemcpyDeviceToHost));

	for (unsigned int i=0;i<src_rows;++i)
	{
		for (unsigned int j=0;j<src_cols;++j)
		{
			result[i*src_cols+j]=h_result[i*fft_w+j];
		}
	}

	cufftSafeCall(hipfftDestroy(fftplan_fwd));
	cufftSafeCall(hipfftDestroy(fftplan_inv));

	cutilSafeCall(hipFree(d_src));
	cutilSafeCall(hipFree(d_kernel));
	cutilSafeCall(hipFree(d_padded_src));
	cutilSafeCall(hipFree(d_padded_kernel));
	cutilSafeCall(hipFree(d_src_spectrum));
	cutilSafeCall(hipFree(d_kernel_spectrum));
	free(h_result);
}

void convolution2DBankCu(const float* src,
	const unsigned int src_rows,
	const unsigned int src_cols,
	float** kernels,
	const unsigned int kernels_num,
	const unsigned int kernel_rows,
	const unsigned int kernel_cols,
	float** result)
{
	const int kernel_h = kernel_rows;
	const int kernel_w = kernel_cols;
	const int data_h = src_rows;
	const int data_w = src_cols;
	const int fft_h = snapTransformSize(data_h + kernel_h - 1);
	const int fft_w = snapTransformSize(data_w + kernel_w - 1);

	float* h_result;
	h_result = (float*)malloc(sizeof(float) * fft_h * fft_w);

	float* d_src;
	float* d_kernel;

	float* d_padded_src;
	float* d_padded_kernel;
	float* d_padded_temp;

	fComplex* d_src_spectrum;
	fComplex* d_kernel_spectrum;
	fComplex* d_temp_spectrum;

	//allocate some space
	cutilSafeCall(hipMalloc((void**)&d_src,data_h * data_w * sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_kernel,kernel_h * kernel_w * sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_padded_src,fft_h * fft_w * sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_padded_kernel,fft_h * fft_w * sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_padded_temp,fft_h * fft_w * sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_src_spectrum,fft_h * (fft_w / 2 + 1)*sizeof(fComplex)));
	cutilSafeCall(hipMalloc((void**)&d_kernel_spectrum,fft_h * (fft_w / 2 + 1)*sizeof(fComplex)));
	cutilSafeCall(hipMalloc((void**)&d_temp_spectrum,fft_h * (fft_w / 2 + 1)*sizeof(fComplex)));

	cutilSafeCall(hipMemset(d_padded_kernel,0,fft_w * fft_h * sizeof(float)));
	cutilSafeCall(hipMemset(d_padded_src,0,fft_w * fft_h * sizeof(float)));

	//copy image data from host to device
	cutilSafeCall(hipMemcpy(d_src,src,data_h * data_w * sizeof(float),hipMemcpyHostToDevice));
	//pad some additional data(how to pad is very important)
	padData(d_padded_src,fft_h,fft_w,d_src,data_h,data_w,kernel_h / 2,kernel_w / 2);

	hipfftHandle fftplan_fwd,fftplan_inv;
	cufftSafeCall(hipfftPlan2d(&fftplan_fwd,fft_h,fft_w,HIPFFT_R2C));
	cufftSafeCall(hipfftPlan2d(&fftplan_inv,fft_h,fft_w,HIPFFT_C2R));

	//compute spectrum of image
	cufftSafeCall(hipfftExecR2C(fftplan_fwd,(hipfftReal*)d_padded_src,(hipfftComplex*)d_src_spectrum));

	for (int kernel_index = 0; kernel_index < int(kernels_num); ++kernel_index)
	{
		//copy kernel data from host to device
		cutilSafeCall(hipMemcpy(d_kernel,kernels[kernel_index],kernel_h*kernel_w*sizeof(float),hipMemcpyHostToDevice));
		//pad some additional data
		padData(d_padded_kernel,fft_h,fft_w,d_kernel,kernel_h,kernel_w);

		//compute spectrum of kernel
		cufftSafeCall(hipfftExecR2C(fftplan_fwd,(hipfftReal*)d_padded_kernel,(hipfftComplex*)d_kernel_spectrum));

		//compute convolution
		cutilSafeCall(cutilDeviceSynchronize());
		conjugate(d_kernel_spectrum,fft_h * (fft_w / 2 + 1));
		multiplyAndScale(d_src_spectrum,d_kernel_spectrum,1.0f / (fft_h * fft_w),d_temp_spectrum,fft_h * (fft_w / 2 + 1));


		cufftSafeCall(hipfftExecC2R(fftplan_inv,(hipfftComplex*)d_temp_spectrum,(hipfftReal*)d_padded_temp));
		cutilSafeCall(cutilDeviceSynchronize());

		cutilSafeCall(hipMemcpy(h_result,d_padded_temp,fft_h * fft_w * sizeof(float),hipMemcpyDeviceToHost));

		for (unsigned int i = 0; i < src_rows; ++i)
		{
			for (unsigned int j = 0; j < src_cols; ++j)
			{
				result[kernel_index][i * src_cols + j] = h_result[i * fft_w + j];
			}
		}
	}

	cufftSafeCall(hipfftDestroy(fftplan_fwd));
	cufftSafeCall(hipfftDestroy(fftplan_inv));

	cutilSafeCall(hipFree(d_src));
	cutilSafeCall(hipFree(d_kernel));
	cutilSafeCall(hipFree(d_padded_src));
	cutilSafeCall(hipFree(d_padded_kernel));
	cutilSafeCall(hipFree(d_padded_temp));
	cutilSafeCall(hipFree(d_src_spectrum));
	cutilSafeCall(hipFree(d_kernel_spectrum));
	cutilSafeCall(hipFree(d_temp_spectrum));
	free(h_result);
}

void matchTemplateCu(const float* match_target,
	const unsigned int target_rows,
	const unsigned int target_cols,
	float* match_template,
	unsigned int template_rows,
	unsigned int template_cols,
	float* similarity)
{

}

//for the cuda 4.2
void matchTemplateSQDIFFCu(const float* match_target,
	unsigned int target_rows,
	unsigned int target_cols,
	const float* match_template,
	unsigned int template_rows,
	unsigned int template_cols,
	float* similarity)
{
	// 		//some preprocess for match_template image
	// 		const int kernel_h=template_rows;
	// 		const int kernel_w=template_cols;
	// 		const int data_h=target_rows;
	// 		const int data_w=target_cols;
	// 		const int fft_h=snapTransformSize(data_h+kernel_h-1);
	// 		const int fft_w=snapTransformSize(data_w+kernel_w-1);
	// 
	// 		float* h_similarity;
	// 		h_similarity=(float*)malloc(sizeof(float)*fft_h*fft_w);
	// 
	// 		float* d_match_target;
	// 		float* d_match_template;
	// 
	// 		float* d_padded_target;
	// 		float* d_padded_template;
	// 		
	// 		fComplex* d_target_spectrum;
	// 		fComplex* d_template_spectrum;
	// 
	// 		cutilSafeCall(hipMalloc((void**)&d_match_target,data_h*data_w*sizeof(float)));
	// 		cutilSafeCall(hipMalloc((void**)&d_match_template,kernel_h*kernel_w*sizeof(float)));
	// 
	// 		cutilSafeCall(hipMalloc((void**)&d_padded_target,fft_h*fft_w*sizeof(float)));
	// 		cutilSafeCall(hipMalloc((void**)&d_padded_template,fft_h*fft_w*sizeof(float)));
	// 		
	// 		cutilSafeCall(hipMalloc((void**)&d_target_spectrum,fft_h * (fft_w / 2 + 1)*sizeof(fComplex)));
	// 		cutilSafeCall(hipMalloc((void**)&d_template_spectrum,fft_h * (fft_w / 2 + 1)*sizeof(fComplex)));
	// 
	// 		cutilSafeCall(hipMemcpy(d_match_template,match_template,kernel_h*kernel_w*sizeof(float),hipMemcpyHostToDevice));
	// 		cutilSafeCall(hipMemcpy(d_match_target,match_target,data_h*data_w*sizeof(float),hipMemcpyHostToDevice));
	// 
	// 		cutilSafeCall(hipMemset(d_padded_template,0,fft_w*fft_h*sizeof(float)));
	// 		cutilSafeCall(hipMemset(d_padded_target,0,fft_w*fft_h*sizeof(float)));
	// 
	// 		padData(d_padded_target,fft_h,fft_w,d_match_target,data_h,data_w);
	// 		padData(d_padded_template,fft_h,fft_w,d_match_template,kernel_h,kernel_w);
	// 
	// 		hipfftHandle fftplan_fwd,fftplan_inv;
	// 		cufftSafeCall(hipfftPlan2d(&fftplan_fwd,fft_h,fft_w,HIPFFT_R2C));
	// 		cufftSafeCall(hipfftPlan2d(&fftplan_inv,fft_h,fft_w,HIPFFT_C2R));
	// 
	// 		//running GPU FFT convolution
	// 		cufftSafeCall(hipfftExecR2C(fftplan_fwd,(hipfftReal*)d_padded_target,(hipfftComplex*)d_target_spectrum));
	// 		cufftSafeCall(hipfftExecR2C(fftplan_fwd,(hipfftReal*)d_padded_template,(hipfftComplex*)d_template_spectrum));
	// 
	// 		cutilSafeCall(cutilDeviceSynchronize());
	//  		conjugate(d_template_spectrum,fft_h * (fft_w / 2 + 1));
	//  		multiplyAndScale(d_target_spectrum,d_template_spectrum,1.0/(fft_h*fft_w),d_target_spectrum,fft_h * (fft_w / 2 + 1));
	// 
	// 
	// 		cufftSafeCall(hipfftExecC2R(fftplan_inv,(hipfftComplex*)d_target_spectrum,(hipfftReal*)d_padded_target));
	// 		cutilSafeCall(cutilDeviceSynchronize());
	// 		
	// 		cutilSafeCall(hipMemcpy(h_similarity,d_padded_target,fft_h*fft_w*sizeof(float),hipMemcpyDeviceToHost));
	// 	
	// 		for (unsigned int i=0;i<target_rows;++i)
	// 		{
	// 			for (unsigned int j=0;j<target_cols;++j)
	// 			{
	// 				similarity[i*target_cols+j]=h_similarity[i*fft_w+j];
	// 			}
	// 		}
	// 
	// 		cufftSafeCall(hipfftDestroy(fftplan_fwd));
	// 		cufftSafeCall(hipfftDestroy(fftplan_inv));
	// 
	// 		cutilSafeCall(hipFree(d_match_target));
	// 		cutilSafeCall(hipFree(d_match_template));
	// 		cutilSafeCall(hipFree(d_padded_target));
	// 		cutilSafeCall(hipFree(d_padded_template));
	// 		cutilSafeCall(hipFree(d_target_spectrum));
	// 		cutilSafeCall(hipFree(d_template_spectrum));
	// 		free(h_similarity);

	convolution2DCu(match_target,target_rows,target_cols,
		match_template,template_rows,template_cols,
		similarity);

	//compute term 2
	//compute integral image
	float* sq_match_target=(float*)malloc(sizeof(float)*target_rows*target_cols);
	float* sq_integral=(float*)malloc(sizeof(float)*target_rows*target_cols);

	for (unsigned int i=0;i<target_rows;++i)
	{
		const float* row_match_target_data=match_target+i*target_cols;
		float* row_sq_match_target_data=sq_match_target+i*target_cols;

		for (unsigned int j=0;j<target_cols;++j)
		{
			row_sq_match_target_data[j]=row_match_target_data[j]*row_match_target_data[j];
		}
	}

	integralImage(sq_match_target,sq_integral,target_rows,target_cols);

	float* term2=(float*)malloc(sizeof(float)*target_rows*target_cols);
	for (unsigned int i=0;i<target_rows;++i)
	{
		float* row_term2_data=term2+i*target_cols;

		for (unsigned int j=0;j<target_cols;++j)
		{
			unsigned int extend_r=EAGLEEYE_MIN((i+template_rows),(target_rows-1));
			unsigned int extend_c=EAGLEEYE_MIN((j+template_cols),(target_cols-1));

			row_term2_data[j]=sq_integral[extend_r*target_cols+extend_c]-
				sq_integral[i*target_cols+extend_c]-
				sq_integral[extend_r*target_cols+j]+
				sq_integral[i*target_cols+j];
		}
	}

	//compute term3
	float term3=0;
	for (unsigned int i=0;i<template_rows;++i)
	{
		const float* row_template_data=match_template+i*template_cols;

		for (unsigned int j=0;j<template_cols;++j)
		{
			term3+=row_template_data[j]*row_template_data[j];
		}
	}

	for (unsigned int i=0;i<target_rows;++i)
	{
		float* row_similarity_data=similarity+i*target_cols;

		float* row_term2_data=term2+i*target_cols;

		for (unsigned int j=0;j<target_cols;++j)
		{
			row_similarity_data[j]=row_term2_data[j]-2*row_similarity_data[j]+term3;
		}
	}

	free(sq_integral);
	free(sq_match_target);
	free(term2);
}

void integralImage(float* imagedata,float* sum,unsigned int image_h,unsigned int image_w)
{
	memset(sum,0,sizeof(float)*image_h*image_w);

	for (unsigned int i=0;i<image_h;++i)
	{
		float* row_data_ptr=imagedata+i*image_w;
		float* row_sum_data_ptr=sum+i*image_w;

		float cii=0;
		if (i==0)
		{
			for (unsigned int j=0;j<image_w;++j)
			{
				if (j==0)
				{
					cii=row_data_ptr[j];
				}
				else
				{
					cii=cii+row_data_ptr[j];
				}

				row_sum_data_ptr[j]=cii;
			}
		}
		else
		{
			float* row_sum_up_data_ptr=sum+(i-1)*image_w;
			for (unsigned int j=0;j<image_w;++j)
			{
				if (j==0)
				{
					cii=row_data_ptr[j];
				}
				else
				{
					cii=cii+row_data_ptr[j];
				}

				row_sum_data_ptr[j]=row_sum_up_data_ptr[j]+cii;
			}
		}
	}
}
}
