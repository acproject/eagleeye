#include "hip/hip_runtime.h"
#include "commoncu.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <cutil_inline.h>

namespace eagleeye
{
bool checkCUDAProfile(int dev, int min_runtime_ver, int min_compute)
{
	int runtime_ver = 0;     

	hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, dev);

	fprintf(stderr,"\nDevice %d: \"%s\"\n", dev, device_prop.name);
	hipRuntimeGetVersion(&runtime_ver);

	if (min_runtime_ver>runtime_ver||min_compute>device_prop.major)
	{
		fprintf(stderr,"  CUDA Runtime Version     :\t%d.%d\n", runtime_ver/1000, (runtime_ver%100)/10);
		fprintf(stderr,"  CUDA Compute Capability  :\t%d.%d\n", device_prop.major, device_prop.minor);
		return false;
	}

	return true;
}

int findCapableDevice(int argc,char **argv)
{
	int device_count=0;
	hipError_t error_id=hipGetDeviceCount(&device_count);
	if (error_id!=hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n->%s\n",(int)error_id,hipGetErrorString(error_id));
		return -1;
	}

	if (device_count==0)
	{
		fprintf(stderr,"There is no device supporting CUDA.\n");
		return -1;
	}
	else
	{
		fprintf(stderr,"Found %d CUDA Capable Device(s).\n",device_count);
	}

	int best_dev=-1;
	hipDeviceProp_t best_device_prop;
	for (int dev=0;dev<device_count;++dev)
	{
		hipDeviceProp_t device_prop;
		hipGetDeviceProperties(&device_prop,dev);

		if ((best_dev==-1)||(best_device_prop.major<device_prop.major))
		{
			best_dev=dev;
			best_device_prop=device_prop;
		}
	}

	if (best_dev!=-1)
	{
		fprintf(stderr,"Setting active device to %d\n",best_dev);
	}

	return best_dev;
}

bool iniCuda(int argc,char ** argv)
{
	int dev=findCapableDevice(argc,argv);
	if (dev!=-1)
	{
		hipSetDevice(dev);
		return true;
	}
	else
		return false;
}

//////////////////////////////////////////////////////////////////////////
__global__ void conjugate_kernel(fComplex* d_data,unsigned int count)
{
	const unsigned int x=blockDim.x*blockIdx.x+threadIdx.x;

	if (x<count)
	{
		d_data[x].y=-d_data[x].y;
	}
}

void conjugate(fComplex* d_data,unsigned int count)
{
	dim3 threads(256,1);
	dim3 grid(iDivUp(count,threads.x),1);	

	conjugate_kernel<<<grid,threads>>>(d_data,count);
}

//////////////////////////////////////////////////////////////////////////

__global__ void subtractScalar_kernel(float* d_data,unsigned int d_h,unsigned int d_w,float value)
{
	const unsigned int y=blockDim.y*blockIdx.y+threadIdx.y;
	const unsigned int x=blockDim.x*blockIdx.x+threadIdx.x;

	if (y<d_h&&x<d_w)
	{
		d_data[y*d_h+x]-=value;
	}
}

void subtractScalar(float* d_data,unsigned int d_h,unsigned int d_w,float value)
{
	dim3 threads(32,8);
	dim3 grid(iDivUp(d_w,threads.x),iDivUp(d_h,threads.y));

	subtractScalar_kernel<<<grid,threads>>>(d_data,d_h,d_w,value);
}

//////////////////////////////////////////////////////////////////////////

__global__ void multiply_kernel(fComplex* a,fComplex* b,fComplex* c,unsigned int count)
{
	const unsigned int x=blockDim.x*blockIdx.x+threadIdx.x;

	if (x<count)
	{
		unsigned int index=x;
		float real_part,image_part;

		real_part=a[index].x*b[index].x-a[index].y*b[index].y;
		image_part=a[index].x*b[index].y+b[index].x*a[index].y;

		c[index].x=real_part;
		c[index].y=image_part;
	}
}

void multiply(fComplex* d_multiply_term1,fComplex* d_multiply_term2,fComplex* d_result,unsigned int count)
{
	dim3 threads(256,1);
	dim3 grid(iDivUp(count,threads.x),1);

	multiply_kernel<<<grid,threads>>>(d_multiply_term1,d_multiply_term2,d_result,count);
}

//////////////////////////////////////////////////////////////////////////
__global__ void multiplyAndScale_kernel(fComplex* a,fComplex* b,float scale,fComplex* c,unsigned int count)
{
	const unsigned int x=blockDim.x*blockIdx.x+threadIdx.x;

	if (x<count)
	{
		unsigned int index=x;
		float real_part,image_part;

		real_part=a[index].x*b[index].x-a[index].y*b[index].y;
		image_part=a[index].x*b[index].y+b[index].x*a[index].y;

		c[index].x=real_part*scale;
		c[index].y=image_part*scale;
	}
}

void multiplyAndScale(fComplex* a,fComplex* b,float scale,fComplex* c,unsigned int count)
{
	dim3 threads(256,1);
	dim3 grid(iDivUp(count,threads.x),1);

	multiplyAndScale_kernel<<<grid,threads>>>(a,b,scale,c,count);
}

//////////////////////////////////////////////////////////////////////////
__global__ void padData_kernel(float* d_dst,unsigned int dst_h,unsigned int dst_w,
	float* d_src,unsigned int src_h,unsigned int src_w,int offset_h,int offset_w)
{
	const unsigned int y=blockDim.y*blockIdx.y+threadIdx.y;
	const unsigned int x=blockDim.x*blockIdx.x+threadIdx.x;

	if (y<src_h&&x<src_w)
	{
		d_dst[(y+offset_h)*dst_w+x+offset_w]=LOAD_FLOAT(y*src_w+x);
	}
}

void padData(float *d_dst,unsigned int dst_h,unsigned int dst_w,float *d_src,unsigned int src_h,unsigned int src_w,int offset_h,int offset_w)
{
	dim3 threads(32, 8);
	dim3 grid(iDivUp(src_w, threads.x), iDivUp(src_h, threads.y));

	SET_FLOAT_BASE;

	padData_kernel<<<grid,threads>>>(d_dst,dst_h,dst_w,d_src,src_h,src_w,offset_h,offset_w);
}
}
