#include "hip/hip_runtime.h"
#include "stdafx.h"
//#include "hip/hip_runtime.h"

#include "DragBarEnhanceCu_kernel.h"

__global__ void BuLiCUDA(WORDTYPE* Tab1,float k,float db)
{
	int nx=blockIdx.x * blockDim.x + threadIdx.x;

	if(nx<(MAX_GRAY+1))
	{
		float pix;
		pix=nx*k+db;  
		if(pix<0) Tab1[nx]=0;
		else if(pix>MAX_GRAY) Tab1[nx]=MAX_GRAY;
		else Tab1[nx]=(WORDTYPE)pix;
	}

}


__global__ void HiCUDA(const WORDTYPE* pOrg, WORDTYPE* pRes,int h, int w,WORDTYPE* temp1,UINT unMovebyte)
{
	int nx=blockIdx.x * blockDim.x + threadIdx.x;
	int ny=blockIdx.y * blockDim.y + threadIdx.y;
	int nindex = ny * w + nx;

	if(nx<w && ny<h)
		pRes[nindex] = temp1[pOrg[nindex]>>unMovebyte]<<unMovebyte;

}


__global__ void DeCUDA(const WORDTYPE* pOrg, WORDTYPE* pRes,float detail,int h, int w, DetailPara depara,UINT unMovebyte)
{
	int nx=blockIdx.x * blockDim.x + threadIdx.x;
	int ny=blockIdx.y * blockDim.y + threadIdx.y;
	int nindex = ny * w + nx;

	WORDTYPE wBackgroundThreshold = depara.wBackgroundThreshold;
	int nRadius = depara.nRadius;
	float fDetailDegree = depara.fDetailDegree;
	WORDTYPE nGrayThreshold = depara.nGrayThreshold;
	float fDifThreshold = depara.fDifThreshold;
	float fEdgeDegree = depara.fEdgeDegree;

	if(nx<w && ny<h)
	{
		double dTempDouble,dMean,sum;
		int r,s;
		float expIndex;
		float dif;
		float curGray;

		if(nx<3 || ny<3 || nx>w-4 || ny>h-4)//��Ե����
		{
			pRes[nindex] = pOrg[nindex]>>unMovebyte;
		}
		else
		{
			if((pOrg[nindex]>>unMovebyte) < wBackgroundThreshold>>unMovebyte)
			{
    		 	for(r = -nRadius; r<=nRadius; r++)
		    	{
				    for(s = -nRadius;s<=nRadius; s++)
				    {
				    	sum += (pOrg[nindex + r*w + s]>>unMovebyte);
				    }
			    }
			    dMean = (WORDTYPE)(sum/((2*nRadius+1)*(2*nRadius+1)));
		    	dTempDouble = (pOrg[nindex]>>unMovebyte)-dMean;

			    if(detail>0)
				{
					if(detail<0.5)
					{
						fDetailDegree=detail*10+5;
						fEdgeDegree=detail*4+1;
					}
					else if(detail<0.8)
					{
						fDetailDegree=detail*15+2.5;
						fEdgeDegree=detail*6;
					}
					else 
					{
						fDetailDegree=detail*20-1.5;
						fEdgeDegree=detail*8-1.6;
					}	

					if(((pOrg[nindex]>>unMovebyte)-dMean)>=0)
						expIndex=3*detail*fDetailDegree;//��ϸ����ǿ�̶�
					else
						expIndex=4*detail*fDetailDegree;//��ϸ����ǿ�̶�

					if((pOrg[nindex]>>unMovebyte)>nGrayThreshold)//�Ҷ���ֵ���Ҷȸ���һ������ֵ���򲻽���ϸ����ǿ��ֻ��΢���б�Ե��ǿ
					{
						dTempDouble=(pOrg[nindex]>>unMovebyte)+((pOrg[nindex]>>unMovebyte)-dMean)*3; 
					}
					else
					{
						curGray = (float)(pOrg[nindex]>>unMovebyte)/MAX_GRAY;
						dif	= curGray - (float)dMean/MAX_GRAY;
					    if (fabs(dif)*curGray*curGray*curGray < fDifThreshold)  //0.002)  //0.0005)	
						{
							dif	*= 2;          //cos����x����ѹ��������Ĭ��ȡ1�����ǲ�����ѹ��
							dif = min(dif,1.0f);
							if (dif>=0)
							{	
								dTempDouble = (pOrg[nindex]>>unMovebyte) + expIndex* cos(3.14159/2*dif) * MAX_GRAY * 0.001 + ((pOrg[nindex]>>unMovebyte)-dMean)*4*fEdgeDegree*detail;                //����Ե��ǿ�̶�							
							}
							else
							{
								dTempDouble = (pOrg[nindex]>>unMovebyte) - expIndex* cos(3.14159/2*dif) * MAX_GRAY * 0.001 + ((pOrg[nindex]>>unMovebyte)-dMean)*8*fEdgeDegree*detail;                //����Ե��ǿ�̶�
							}
						}
						else
						{
							dif	*= 2;          //cos����x����ѹ��������Ĭ��ȡ1�����ǲ�����ѹ��
							dif = min(dif,1.0f);
							if (dif>=0)
							{
								dTempDouble =(pOrg[nindex]>>unMovebyte) +  expIndex * cos(3.14159/2*dif) * MAX_GRAY * 0.01 + ((pOrg[nindex]>>unMovebyte)-dMean)*4*fEdgeDegree*detail; 							
							}
							else
							{
								dTempDouble= (pOrg[nindex]>>unMovebyte) - expIndex* cos(3.14159/2*dif) * MAX_GRAY * 0.01 + ((pOrg[nindex]>>unMovebyte)-dMean)*8*fEdgeDegree*detail; 	
							}
						}		
					}
				}
				else
				{
					dTempDouble=(pOrg[nindex]>>unMovebyte)+dTempDouble*detail;
				}
			}
			else 
			{
				dTempDouble=(pOrg[nindex]>>unMovebyte);

			}

			if(dTempDouble<0) dTempDouble = 0;
			if(dTempDouble>MAX_GRAY) dTempDouble = MAX_GRAY;
			pRes[nindex] = ((WORDTYPE)dTempDouble)<<unMovebyte;
		}
	}

}
