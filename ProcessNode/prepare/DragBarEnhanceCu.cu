#include "hip/hip_runtime.h"
#include "stdafx.h"
//#include "hip/hip_runtime.h"

#include "DragBarEnhanceCu_kernel.h"
#include "DragBarEnhanceCu.h"

extern UINT g_unMovebyte;

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

BOOL InitCUDA(void){return true;}

#else

BOOL InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return FALSE;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1 && prop.major != 9999) {                //�ҵ����õ��Կ���ֹͣ������ ����ж���Կ���������򣬿���Э������
				break;
			}
		}
	}

	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");   // ���û�п�֧��cuda���Կ�����Ӧ����cpu�㷨
		return FALSE;
	}
	hipSetDevice(i);  

	printf("CUDA initialized.\n");

	return TRUE;
}

#endif


bool LigConProcCuda(const WORDTYPE* pOrg, WORDTYPE* pRes, int w,int h, float* fBarPos,bool bIsDrag,bool bIsROI,RECT* mROIRect)
{
	//��ʼ������
    float light=fBarPos[0],contrast=fBarPos[1];
	float k,nx,ny,db,nx1,ny1;
	int i,nStart,nStep,nTabLen=0;

#ifdef ISWORD
	nTabLen=512;
	nStart=50;
	nStep=10;
#else
	nTabLen=1000;
	nStart=800;
	nStep=160;
#endif

	WORDTYPE *pBarTab=new WORDTYPE[nTabLen];//�̶�ת����
	pBarTab[0] = nStart;
	for(i=1;i<nTabLen;i++)
	{
		if(pBarTab[i-1]+nStep>=MAX_GRAY)
		{
			pBarTab[i] = MAX_GRAY;
			break;
		}
		else
			pBarTab[i] = pBarTab[i-1]+nStep;
#ifdef ISWORD
		nStep++;
#else
		nStep+=2;
#endif
	}
	nTabLen = i;
	int newlight=pBarTab[(int)((1-fabs(light))*nTabLen)];  //�̶ȱ任

	ny=(float)MAX_GRAY/2;
	contrast *= 0.95;

	if(light>=0)
	{
		ny1=MAX_GRAY;
		nx=(float)newlight/2;
		if(contrast>=0)
		{
			nx1=nx*(1-contrast)+nx;
			ny1=MAX_GRAY;
		}
		else
		{
			ny1=-ny*contrast;
			nx1=0;
		}
	}
	else
	{
		nx=MAX_GRAY-(float)newlight/2;
		if(contrast>=0)
		{
			nx1=MAX_GRAY-contrast*newlight/2;
			ny1=MAX_GRAY;
		}
		else
		{
			ny1=MAX_GRAY-contrast*ny;
			nx1=MAX_GRAY;
		}
	}
	k=(ny-ny1)/(nx-nx1);
	db=ny-nx*k;

	//����GPU�������ȺͶԱȶ�
	//����
	WORDTYPE *LiconTable = new WORDTYPE[MAX_GRAY+1];
	memset(LiconTable,0,sizeof(WORDTYPE) * (MAX_GRAY+1));
	WORDTYPE * TabResult1;
	hipMalloc((void**) &TabResult1, sizeof(WORDTYPE) * (MAX_GRAY+1));

	BuLiCUDA <<<(MAX_GRAY+1)/64,64>>>(TabResult1,k,db);//��������
	hipDeviceSynchronize();//ͬ�����ȴ����н����������
	hipMemcpy(LiconTable, TabResult1, sizeof(WORDTYPE) * (MAX_GRAY+1), hipMemcpyDeviceToHost);

	//���
	WORDTYPE * pDeviceOrignal;
	WORDTYPE * pDeviceResult;
	WORDTYPE * temp1;
	hipMalloc((void**) &temp1, sizeof(WORDTYPE) * (MAX_GRAY+1));
	hipMalloc((void**) &pDeviceOrignal, sizeof(WORDTYPE) * h*w);
	hipMalloc((void**) &pDeviceResult, sizeof(WORDTYPE) * h*w);
	hipMemcpy(temp1, LiconTable, sizeof(WORDTYPE) * (MAX_GRAY+1), hipMemcpyHostToDevice);
	hipMemcpy(pDeviceOrignal, pOrg, sizeof(WORDTYPE) * h*w, hipMemcpyHostToDevice);

	dim3 dg((w+15)/16,(h+15)/16),dbb(16,16);
	HiCUDA <<<dg,dbb>>>(pDeviceOrignal,pDeviceResult,h,w,temp1,g_unMovebyte);//��������
	hipDeviceSynchronize();//ͬ�����ȴ����н����������
	hipMemcpy(pRes, pDeviceResult, sizeof(WORDTYPE) * h*w, hipMemcpyDeviceToHost);

	hipFree(pDeviceResult);
	hipFree(pDeviceOrignal);
	hipFree(temp1);
	hipFree(TabResult1);
	delete[] pBarTab;
	delete[] LiconTable;
	return TRUE;
}

bool DetailProcCuda(const WORDTYPE* pOrg, WORDTYPE* pRes, int w,int h, float* fBarPos,bool bIsDrag,bool bIsROI,RECT* mROIRect,DetailPara depara)
{
	//���������þֲ���ǿ�ķ�����ѡ��2*r-1��*��2*r-1������ķ�Χ�ľ�ֵ��Ϊ�����Ƿ���б�Ե��ǿ������
    float detail=fBarPos[0];
	//����GPU����ϸ����ǿ
	WORDTYPE * pDeviceOrignal;
	WORDTYPE * pDeviceResult;
	hipMalloc((void**) &pDeviceOrignal, sizeof(WORDTYPE) * h*w);
	hipMalloc((void**) &pDeviceResult, sizeof(WORDTYPE) * h*w);
	hipMemcpy(pDeviceOrignal, pOrg, sizeof(WORDTYPE) * h*w, hipMemcpyHostToDevice);

	dim3 dg((w+15)/16,(h+15)/16),db(16,16);
	DeCUDA <<<dg,db>>>(pDeviceOrignal,pDeviceResult,detail,h,w,depara,g_unMovebyte);//��������
	hipDeviceSynchronize();//ͬ�����ȴ����н����������
	hipMemcpy(pRes, pDeviceResult, sizeof(WORDTYPE) * h*w, hipMemcpyDeviceToHost);

	hipFree(pDeviceResult);
	hipFree(pDeviceOrignal);

	return TRUE;

}